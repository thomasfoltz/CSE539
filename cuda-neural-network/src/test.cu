#include <chrono>
#include <fstream>
#include <vector>
#include <iostream>
#include <time.h>

#include "neural_network.hh"
#include "layers/linear_layer.hh"
#include "layers/relu_activation.hh"
#include "layers/sigmoid_activation.hh"
#include "nn_utils/nn_exception.hh"
#include "nn_utils/bce_cost.hh"
#include "nn_utils/matrix.hh"

#include "coordinates_dataset.hh"

#define total_coordinates 32768

float computeAccuracy(const Matrix& predictions, const Matrix& targets);

void loadModel(NeuralNetwork& nn, const char* filePath) {
    std::string item_name;
    std::ifstream nameFileout;
    nameFileout.open(filePath);
    std::string line;
    while(std::getline(nameFileout, line))
    {
        if(line.compare("Linear") == 0){
            std::string shapeX, shapeY;
            std::getline(nameFileout, shapeX);
            std::getline(nameFileout, shapeY);

            int xW = std::stoi(shapeX);
            int yW = std::stoi(shapeY);

            float* weights = new float[xW*yW];
            
            std::string number;

            for(int i = 0; i < xW*yW; i++){
                std::getline(nameFileout, number);
                weights[i] = stof(number);
            }

            std::getline(nameFileout, shapeX);
            std::getline(nameFileout, shapeY);

            int x = std::stoi(shapeX);
            int y = std::stoi(shapeY);

            float* bias = new float[x*y];
            
            for(int i = 0; i < x*y; i++){
                std::getline(nameFileout, number);
                bias[i] = stof(number);
            }
                                                        

            nn.addLayer(new LinearLayer("linear", Shape(xW, yW), weights, bias));
        } 
        else if(line.compare("Sigmoid") == 0){
	        nn.addLayer(new SigmoidActivation("sigmoid"));
        }
        else if(line.compare("Relu") == 0){
	        nn.addLayer(new ReLUActivation("relu"));
        }
    }
}
 
int main(int argc, char** argv) {
    NeuralNetwork nn1, nn2, nn3, nn4;
    loadModel(nn1, "nn1.txt");
    loadModel(nn2, "nn2.txt");
    loadModel(nn3, "nn3.txt");
    loadModel(nn4, "nn4.txt");

    int number_of_streams = std::stoi(argv[1]);
    int batch_size = std::stoi(argv[2]);
    int number_of_batches = total_coordinates/batch_size;

    srand(1000);
    CoordinatesDataset dataset(batch_size, number_of_batches);
    Matrix Y1, Y2, Y3, Y4;

    if (number_of_streams == 1) {
        hipStream_t stream1; 
        hipStreamCreate (&stream1);
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < number_of_batches; i++) {
            Y1 = nn1.forward(dataset.getBatches().at(i), stream1);
            Y2 = nn2.forward(dataset.getBatches().at(i), stream1);
            Y3 = nn3.forward(dataset.getBatches().at(i), stream1);
            Y4 = nn4.forward(dataset.getBatches().at(i), stream1);
            Y1.copyDeviceToHost();
            Y2.copyDeviceToHost();
            Y3.copyDeviceToHost();
            Y4.copyDeviceToHost();
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Total time: " << duration.count() << " milliseconds" << std::endl;
    }
    else if (number_of_streams == 2) {
        hipStream_t stream1, stream2;
        hipStreamCreate (&stream1);
        hipStreamCreate (&stream2);
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < number_of_batches; i++) {
            Y1 = nn1.forward(dataset.getBatches().at(i), stream1);
            Y2 = nn2.forward(dataset.getBatches().at(i), stream2);
            Y3 = nn3.forward(dataset.getBatches().at(i), stream1);
            Y4 = nn4.forward(dataset.getBatches().at(i), stream2);
            Y1.copyDeviceToHost();
            Y2.copyDeviceToHost();
            Y3.copyDeviceToHost();
            Y4.copyDeviceToHost();
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Total time: " << duration.count() << " milliseconds" << std::endl;
    }
    else if (number_of_streams == 4) {
        hipStream_t stream1, stream2, stream3, stream4;
        hipStreamCreate (&stream1);
        hipStreamCreate (&stream2);
        hipStreamCreate (&stream3);
        hipStreamCreate (&stream4);
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < number_of_batches; i++) {
            Y1 = nn1.forward(dataset.getBatches().at(i), stream1);
            Y2 = nn2.forward(dataset.getBatches().at(i), stream2);
            Y3 = nn3.forward(dataset.getBatches().at(i), stream3);
            Y4 = nn4.forward(dataset.getBatches().at(i), stream4);
            Y1.copyDeviceToHost();
            Y2.copyDeviceToHost();
            Y3.copyDeviceToHost();
            Y4.copyDeviceToHost();
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Total time: " << duration.count() << " milliseconds" << std::endl;
    }
    else {
        std::cout << "Invalid number of streams" << std::endl;
        return 1;
    }
    
    return 0;
}

float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
	int m = predictions.shape.x;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i]) {
			correct_predictions++;
		}
	}

	return static_cast<float>(correct_predictions) / m;
}
